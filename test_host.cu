#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "test_kernel.hu"
//#include <stdio.h>
int A[4][4];
int B[4][4];
int C[4][4];
int main()
{
 int i,j=0,k; 	
 {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

   int *dev_A;
   int *dev_B;
   
   cudaCheckReturn(hipMalloc((void **) &dev_A, (4) * (4) * sizeof(int)));
   cudaCheckReturn(hipMalloc((void **) &dev_B, (4) * (4) * sizeof(int)));
   
   
   hipBindTexture(NULL,texRef_A, dev_A, (4) * (4) * sizeof(int));
   
   hipBindTexture(NULL,texRef_B, dev_B, (4) * (4) * sizeof(int));
   
   cudaCheckReturn(hipMemcpy(dev_A, A, (4) * (4) * sizeof(int), hipMemcpyHostToDevice));
   cudaCheckReturn(hipMemcpy(dev_B, B, (4) * (4) * sizeof(int), hipMemcpyHostToDevice));
   {
     dim3 k0_dimBlock(4);
     dim3 k0_dimGrid(1);
     kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B);
     cudaCheckKernel();
   }
   
   cudaCheckReturn(hipMemcpy(B, dev_B, (4) * (4) * sizeof(int), hipMemcpyDeviceToHost));
   
   hipUnbindTexture(texRef_A);
   hipUnbindTexture(texRef_B);
   
   cudaCheckReturn(hipFree(dev_A));
   cudaCheckReturn(hipFree(dev_B));
 }
  return 0;
}

