#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "gesummv_org_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gesummv.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gesummv.h"


/* Array initialization. */
static
void init_array(int n,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		DATA_TYPE POLYBENCH_2D(B,N,N,n,n),
		DATA_TYPE POLYBENCH_1D(x,N,n))
{
  int i, j;

  *alpha = 1.5;
  *beta = 1.2;
  for (i = 0; i < n; i++)
    {
      x[i] = (DATA_TYPE)( i % n) / n;
      for (j = 0; j < n; j++) {
	A[i][j] = (DATA_TYPE) ((i*j+1) % n) / n;
	B[i][j] = (DATA_TYPE) ((i*j+2) % n) / n;
      }
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(y,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("y");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, y[i]);
  }
  POLYBENCH_DUMP_END("y");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_gesummv(int n,
		    DATA_TYPE alpha,
		    DATA_TYPE beta,
		    DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		    DATA_TYPE POLYBENCH_2D(B,N,N,n,n),
		    DATA_TYPE POLYBENCH_1D(tmp,N,n),
		    DATA_TYPE POLYBENCH_1D(x,N,n),
		    DATA_TYPE POLYBENCH_1D(y,N,n))
{
  int i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_B;
    double *dev_tmp;
    double *dev_x;
    double *dev_y;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (1300) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (n) * (1300) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_tmp, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_x, (n) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_y, (n) * sizeof(double)));
    
    
    hipBindTexture(NULL,texRef_A, dev_A, (n) * (1300) * sizeof(double));
    
    hipBindTexture(NULL,texRef_B, dev_B, (n) * (1300) * sizeof(double));
    
    hipBindTexture(NULL,texRef_x, dev_x, (n) * sizeof(double));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (1300) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_B, B, (n) * (1300) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x, x, (n) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(ppcg_min(32768, (n + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, alpha, beta, dev_tmp, dev_x, dev_y, n);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(tmp, dev_tmp, (n) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(y, dev_y, (n) * sizeof(double), hipMemcpyDeviceToHost));
    
    hipUnbindTexture(texRef_A);
    hipUnbindTexture(texRef_B);
    hipUnbindTexture(texRef_x);
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_tmp));
    cudaCheckReturn(hipFree(dev_x));
    cudaCheckReturn(hipFree(dev_y));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(tmp, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n, &alpha, &beta,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B),
	      POLYBENCH_ARRAY(x));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gesummv (n, alpha, beta,
		  POLYBENCH_ARRAY(A),
		  POLYBENCH_ARRAY(B),
		  POLYBENCH_ARRAY(tmp),
		  POLYBENCH_ARRAY(x),
		  POLYBENCH_ARRAY(y));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(y)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(tmp);
  POLYBENCH_FREE_ARRAY(x);
  POLYBENCH_FREE_ARRAY(y);

  return 0;
}
