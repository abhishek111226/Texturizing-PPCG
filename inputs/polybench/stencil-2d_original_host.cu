#include <assert.h>
#include <stdio.h>
#include "stencil-2d_original_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* jacobi-2d.c: this file is part of PolyBench/C */


#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "jacobi-2d.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		 DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      {
	A[i][j] = ((DATA_TYPE) i*(j+2) + 2) / n;
	B[i][j] = ((DATA_TYPE) i*(j+3) + 3) / n;
      }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(A,N,N,n,n))

{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("A");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      if ((i * n + j) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
      fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, A[i][j]);
    }
  POLYBENCH_DUMP_END("A");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_jacobi_2d(int tsteps,
			    int n,
			    DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
			    DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
  int t, i, j;

  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (tsteps >= 1 && n >= 3) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_B;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (1300) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (n) * (1300) * sizeof(double)));
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (1300) * sizeof(double), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_B, B, (n) * (1300) * sizeof(double), hipMemcpyHostToDevice));
    for (int c0 = 0; c0 < tsteps; c0 += 1) {
      {
        dim3 k0_dimBlock(16, 32);
        dim3 k0_dimGrid(ppcg_min(256, (n + 30) / 32), ppcg_min(256, (n + 30) / 32));
        kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, tsteps, n, c0);
        cudaCheckKernel();
      }
      
      {
        dim3 k1_dimBlock(16, 32);
        dim3 k1_dimGrid(ppcg_min(256, (n + 30) / 32), ppcg_min(256, (n + 30) / 32));
        kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_B, tsteps, n, c0);
        cudaCheckKernel();
      }
      
    }
    cudaCheckReturn(hipMemcpy(A, dev_A, (n) * (1300) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(B, dev_B, (n) * (1300) * sizeof(double), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int tsteps = TSTEPS;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);


  /* Initialize array(s). */
  init_array (n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_jacobi_2d(tsteps, n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);

  return 0;
}
