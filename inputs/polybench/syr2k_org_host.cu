#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "syr2k_org_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* syr2k.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "syr2k.h"


/* Array initialization. */
static
void init_array(int n, int m,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(C,N,N,n,n),
		DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		DATA_TYPE POLYBENCH_2D(B,N,M,n,m))
{
  int i, j;

  *alpha = 1.5;
  *beta = 1.2;
  for (i = 0; i < n; i++)
    for (j = 0; j < m; j++) {
      A[i][j] = (DATA_TYPE) ((i*j+1)%n) / n;
      B[i][j] = (DATA_TYPE) ((i*j+2)%m) / m;
    }
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
      C[i][j] = (DATA_TYPE) ((i*j+3)%n) / m;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_2D(C,N,N,n,n))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("C");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++) {
	if ((i * n + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
	fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, C[i][j]);
    }
  POLYBENCH_DUMP_END("C");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_syr2k(int n, int m,
		  DATA_TYPE alpha,
		  DATA_TYPE beta,
		  DATA_TYPE POLYBENCH_2D(C,N,N,n,n),
		  DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		  DATA_TYPE POLYBENCH_2D(B,N,M,n,m))
{
  int i, j, k;

//BLAS PARAMS
//UPLO  = 'L'
//TRANS = 'N'
//A is NxM
//B is NxM
//C is NxN
  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (n >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_B;
    double *dev_C;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (n) * (1000) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (n) * (1000) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_C, (n) * (1200) * sizeof(double)));
    
    
    hipBindTexture(NULL,texRef_A, dev_A, (n) * (1000) * sizeof(double));
    
    hipBindTexture(NULL,texRef_B, dev_B, (n) * (1000) * sizeof(double));
    
    hipBindTexture(NULL,texRef_C, dev_C, (n) * (1200) * sizeof(double));
    
    if (m >= 1) {
      cudaCheckReturn(hipMemcpy(dev_A, A, (n) * (1000) * sizeof(double), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_B, B, (n) * (1000) * sizeof(double), hipMemcpyHostToDevice));
    }
    cudaCheckReturn(hipMemcpy(dev_C, C, (n) * (1200) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(ppcg_min(256, (n + 31) / 32), ppcg_min(256, (n + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, dev_C, alpha, beta, n, m);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(C, dev_C, (n) * (1200) * sizeof(double), hipMemcpyDeviceToHost));
    
    hipUnbindTexture(texRef_A);
    hipUnbindTexture(texRef_B);
    hipUnbindTexture(texRef_C);
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_C));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,N,N,n,n);
  POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,N,M,n,m);
  POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,N,M,n,m);

  /* Initialize array(s). */
  init_array (n, m, &alpha, &beta,
	      POLYBENCH_ARRAY(C),
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_syr2k (n, m,
		alpha, beta,
		POLYBENCH_ARRAY(C),
		POLYBENCH_ARRAY(A),
		POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(C)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);

  return 0;
}
