#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "gemm_a_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gemm.c: this file is part of PolyBench/C */


#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gemm.h"


/* Array initialization. */
static
void init_array(int ni, int nj, int nk,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj),
		DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj))
{
  int i, j;

  *alpha = 1;
  *beta = 1;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++)
      C[i][j] = (DATA_TYPE) 0 ;
  for (i = 0; i < ni; i++)
    for (j = 0; j < nk; j++)
      A[i][j] = (DATA_TYPE) (i*ni + j);
  for (i = 0; i < nk; i++)
    for (j = 0; j < nj; j++)
      B[i][j] = (DATA_TYPE) i*ni + j;
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
  int i, j;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("C");
  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
	if ((i * ni + j) % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
	fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, C[i][j]);
    }
  POLYBENCH_DUMP_END("C");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_gemm(int ni, int nj, int nk,
		 DATA_TYPE alpha,
		 DATA_TYPE beta,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj),
		 DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
		 DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj))
{
  int i, j, k;

//BLAS PARAMS
//TRANSA = 'N'
//TRANSB = 'N'
// => Form C := alpha*A*B + beta*C,
//A is NIxNK
//B is NKxNJ
//C is NIxNJ
  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  if (ni >= 1 && nj >= 1) {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    double *dev_A;
    double *dev_B;
    double *dev_C;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (ni) * (1200) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (nk) * (1100) * sizeof(double)));
    cudaCheckReturn(hipMalloc((void **) &dev_C, (ni) * (1100) * sizeof(double)));
    
    
    hipBindTexture(NULL,texRef_C, dev_C, (ni) * (1100) * sizeof(double));
    
    if (nk >= 1) {
      cudaCheckReturn(hipMemcpy(dev_A, A, (ni) * (1200) * sizeof(double), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_B, B, (nk) * (1100) * sizeof(double), hipMemcpyHostToDevice));
    }
    cudaCheckReturn(hipMemcpy(dev_C, C, (ni) * (1100) * sizeof(double), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(ppcg_min(256, (nj + 31) / 32), ppcg_min(256, (ni + 31) / 32));
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, dev_C, alpha, beta, ni, nj, nk);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(C, dev_C, (ni) * (1100) * sizeof(double), hipMemcpyDeviceToHost));
    
    hipUnbindTexture(texRef_C);
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_C));
  }




}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int ni = NI;
  int nj = NJ;
  int nk = NK;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
  POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
  POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);

  /* Initialize array(s). */
  init_array (ni, nj, nk, &alpha, &beta,
	      POLYBENCH_ARRAY(C),
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gemm (ni, nj, nk,
	       alpha, beta,
	       POLYBENCH_ARRAY(C),
	       POLYBENCH_ARRAY(A),
	       POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(ni, nj,  POLYBENCH_ARRAY(C)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(C);
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);

  return 0;
}
