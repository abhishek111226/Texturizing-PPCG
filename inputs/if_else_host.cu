#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "if_else_kernel.hu"
int tex_array[10][10];
//int cangle[360];
int main()
{	
int t, i, j,loop,sum1,sum2,a;
 for(int j=0;j<10;j++)
 for(int i=0;i<10;i++)
 {

	tex_array[i][j]=i+j;
 }
 {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

   int *dev_tex_array;
   
   cudaCheckReturn(hipMalloc((void **) &dev_tex_array, (10) * (10) * sizeof(int)));
   
   
   hipBindTexture(NULL,texRef_tex_array, dev_tex_array, (10) * (10) * sizeof(int));
   
   cudaCheckReturn(hipMemcpy(dev_tex_array, tex_array, (10) * (10) * sizeof(int), hipMemcpyHostToDevice));
   {
     dim3 k0_dimBlock(10, 10);
     dim3 k0_dimGrid(1, 1);
     kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_tex_array);
     cudaCheckKernel();
   }
   
   cudaCheckReturn(hipMemcpy(tex_array, dev_tex_array, (10) * (10) * sizeof(int), hipMemcpyDeviceToHost));
   
   hipUnbindTexture(texRef_tex_array);
   
   cudaCheckReturn(hipFree(dev_tex_array));
 }
 for(int j=0;j<10;j++)
 for(int i=0;i<10;i++)
 {
	sum1+=tex_array[i][j];
 }
  return 0;
}
