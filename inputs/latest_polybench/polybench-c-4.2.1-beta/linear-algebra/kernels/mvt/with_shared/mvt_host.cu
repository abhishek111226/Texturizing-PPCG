#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "mvt_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* mvt.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "mvt.h"


/* Array initialization. */
static
void init_array(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  for (i = 0; i < n; i++)
    {
      x1[i] = (DATA_TYPE) (i % n) / n;
      x2[i] = (DATA_TYPE) ((i + 1) % n) / n;
      y_1[i] = (DATA_TYPE) ((i + 3) % n) / n;
      y_2[i] = (DATA_TYPE) ((i + 4) % n) / n;
      for (j = 0; j < n; j++)
	A[i][j] = (DATA_TYPE) (i*j % n) / n;
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(x1,N,n),
		 DATA_TYPE POLYBENCH_1D(x2,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("x1");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x1[i]);
  }
  POLYBENCH_DUMP_END("x1");

  POLYBENCH_DUMP_BEGIN("x2");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, x2[i]);
  }
  POLYBENCH_DUMP_END("x2");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_mvt(int n,
		DATA_TYPE POLYBENCH_1D(x1,N,n),
		DATA_TYPE POLYBENCH_1D(x2,N,n),
		DATA_TYPE POLYBENCH_1D(y_1,N,n),
		DATA_TYPE POLYBENCH_1D(y_2,N,n),
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n))
{
  int i, j;

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float (*dev_A)[2000];
    float *dev_x1;
    float *dev_x2;
    float *dev_y_1;
    float *dev_y_2;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (2000) * (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_x1, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_x2, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_1, (2000) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_y_2, (2000) * sizeof(float)));
    
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (2000) * (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x1, x1, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x2, x2, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_1, y_1, (2000) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_y_2, y_2, (2000) * sizeof(float), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(63);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_x1, dev_y_1);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(63);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_x2, dev_y_2);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(x1, dev_x1, (2000) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(x2, dev_x2, (2000) * sizeof(float), hipMemcpyDeviceToHost));
    
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_x1));
    cudaCheckReturn(hipFree(dev_x2));
    cudaCheckReturn(hipFree(dev_y_1));
    cudaCheckReturn(hipFree(dev_y_2));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(x1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x2, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_1, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y_2, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_mvt (n,
	      POLYBENCH_ARRAY(x1),
	      POLYBENCH_ARRAY(x2),
	      POLYBENCH_ARRAY(y_1),
	      POLYBENCH_ARRAY(y_2),
	      POLYBENCH_ARRAY(A));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(x1), POLYBENCH_ARRAY(x2)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(x1);
  POLYBENCH_FREE_ARRAY(x2);
  POLYBENCH_FREE_ARRAY(y_1);
  POLYBENCH_FREE_ARRAY(y_2);

  return 0;
}
