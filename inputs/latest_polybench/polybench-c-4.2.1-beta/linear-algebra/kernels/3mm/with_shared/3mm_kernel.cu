#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "3mm_kernel.hu"
__global__ void kernel0(float A[800][1000], float B[1000][900], float E[800][900])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 999; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 999); c4 += 16)
        shared_A[t0][c4] = A[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 999)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 899); c4 += 16)
          shared_B[t0][c4] = B[t0 + c2][32 * b1 + c4];
      __syncthreads();
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 899); c4 += 16) {
        if (c2 == 0)
          E[32 * b0 + t0][32 * b1 + c4] = 0.F;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 999); c5 += 1)
          E[32 * b0 + t0][32 * b1 + c4] += (shared_A[t0][c5] * shared_B[c5][c4]);
      }
      __syncthreads();
    }
}
__global__ void kernel1(float C[900][1200], float D[1200][1100], float F[900][1100])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_C[32][32];
    __shared__ float shared_D[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 1199; c2 += 32) {
      if (32 * b0 + t0 <= 899)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1199); c4 += 16)
          shared_C[t0][c4] = C[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 1199)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16)
          shared_D[t0][c4] = D[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 899)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16) {
          if (c2 == 0)
            F[32 * b0 + t0][32 * b1 + c4] = 0.F;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 1199); c5 += 1)
            F[32 * b0 + t0][32 * b1 + c4] += (shared_C[t0][c5] * shared_D[c5][c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel2(float E[800][900], float F[900][1100], float G[800][1100])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_E[32][32];
    __shared__ float shared_F[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 899; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 899); c4 += 16)
        shared_E[t0][c4] = E[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 899)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16)
          shared_F[t0][c4] = F[t0 + c2][32 * b1 + c4];
      __syncthreads();
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16) {
        if (c2 == 0)
          G[32 * b0 + t0][32 * b1 + c4] = 0.F;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 899); c5 += 1)
          G[32 * b0 + t0][32 * b1 + c4] += (shared_E[t0][c5] * shared_F[c5][c4]);
      }
      __syncthreads();
    }
}
