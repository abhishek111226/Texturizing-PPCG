#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "bicg_kernel.hu"
__global__ void kernel0(float A[2100][1900], float r[2100], float s[1900])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_r[32];
    __shared__ float shared_s[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 2099; c1 += 32) {
        if (t0 + c1 <= 2099)
          shared_r[t0] = r[t0 + c1];
        __syncthreads();
        if (32 * b0 + t0 <= 1899 && c1 == 0)
          shared_s[t0] = 0;
        if (32 * b0 + t0 <= 1899)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 2099); c3 += 1)
            shared_s[t0] = (shared_s[t0] + (shared_r[c3] * A[c1 + c3][32 * b0 + t0]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 1899)
        s[32 * b0 + t0] = shared_s[t0];
    }
}
__global__ void kernel1(float A[2100][1900], float p[1900], float q[2100])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_p[32];
    __shared__ float shared_q[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 1899; c1 += 32) {
        if (t0 + c1 <= 1899) {
          for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 2099); c2 += 1)
            shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
          shared_p[t0] = p[t0 + c1];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 2099 && c1 == 0)
          shared_q[t0] = 0.F;
        if (32 * b0 + t0 <= 2099)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 1899); c3 += 1)
            shared_q[t0] = (shared_q[t0] + (shared_A[t0][c3] * shared_p[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 2099)
        q[32 * b0 + t0] = shared_q[t0];
    }
}
