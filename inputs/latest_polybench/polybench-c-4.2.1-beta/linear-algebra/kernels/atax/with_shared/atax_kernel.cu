#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "atax_kernel.hu"
__global__ void kernel0(float A[1900][2100], float tmp[1900], float x[2100])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_tmp[32];
    __shared__ float shared_x[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 2099; c1 += 32) {
        if (t0 + c1 <= 2099) {
          for (int c2 = 0; c2 <= ppcg_min(31, -32 * b0 + 1899); c2 += 1)
            shared_A[c2][t0] = A[32 * b0 + c2][t0 + c1];
          shared_x[t0] = x[t0 + c1];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 1899 && c1 == 0)
          shared_tmp[t0] = 0.F;
        if (32 * b0 + t0 <= 1899)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 2099); c3 += 1)
            shared_tmp[t0] = (shared_tmp[t0] + (shared_A[t0][c3] * shared_x[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 1899)
        tmp[32 * b0 + t0] = shared_tmp[t0];
    }
}
__global__ void kernel1(float A[1900][2100], float tmp[1900], float y[2100])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_tmp[32];
    __shared__ float shared_y[32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      for (int c1 = 0; c1 <= 1899; c1 += 32) {
        if (t0 + c1 <= 1899)
          shared_tmp[t0] = tmp[t0 + c1];
        __syncthreads();
        if (32 * b0 + t0 <= 2099 && c1 == 0)
          shared_y[t0] = 0;
        if (32 * b0 + t0 <= 2099)
          for (int c3 = 0; c3 <= ppcg_min(31, -c1 + 1899); c3 += 1)
            shared_y[t0] = (shared_y[t0] + (A[c1 + c3][32 * b0 + t0] * shared_tmp[c3]));
        __syncthreads();
      }
      if (32 * b0 + t0 <= 2099)
        y[32 * b0 + t0] = shared_y[t0];
    }
}
