#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "gemm_kernel.hu"
__global__ void kernel0(float A[1000][1200], float B[1200][1100], float C[1000][1100], float alpha, float beta)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 1199; c2 += 32) {
      if (32 * b0 + t0 <= 999)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1199); c4 += 16)
          shared_A[t0][c4] = A[32 * b0 + t0][c2 + c4];
      if (t0 + c2 <= 1199)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16)
          shared_B[t0][c4] = B[t0 + c2][32 * b1 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 999)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16) {
          if (c2 == 0)
            C[32 * b0 + t0][32 * b1 + c4] *= beta;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 1199); c5 += 1)
            C[32 * b0 + t0][32 * b1 + c4] += ((alpha * shared_A[t0][c5]) * shared_B[c5][c4]);
        }
      __syncthreads();
    }
}
