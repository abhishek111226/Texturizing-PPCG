#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "gesummv_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* gesummv.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "gesummv.h"


/* Array initialization. */
static
void init_array(int n,
		DATA_TYPE *alpha,
		DATA_TYPE *beta,
		DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		DATA_TYPE POLYBENCH_2D(B,N,N,n,n),
		DATA_TYPE POLYBENCH_1D(x,N,n))
{
  int i, j;

  *alpha = 1.5;
  *beta = 1.2;
  for (i = 0; i < n; i++)
    {
      x[i] = (DATA_TYPE)( i % n) / n;
      for (j = 0; j < n; j++) {
	A[i][j] = (DATA_TYPE) ((i*j+1) % n) / n;
	B[i][j] = (DATA_TYPE) ((i*j+2) % n) / n;
      }
    }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_1D(y,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("y");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, y[i]);
  }
  POLYBENCH_DUMP_END("y");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_gesummv(int n,
		    DATA_TYPE alpha,
		    DATA_TYPE beta,
		    DATA_TYPE POLYBENCH_2D(A,N,N,n,n),
		    DATA_TYPE POLYBENCH_2D(B,N,N,n,n),
		    DATA_TYPE POLYBENCH_1D(tmp,N,n),
		    DATA_TYPE POLYBENCH_1D(x,N,n),
		    DATA_TYPE POLYBENCH_1D(y,N,n))
{
  int i, j;

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float (*dev_A)[1300];
    float (*dev_B)[1300];
    float *dev_tmp;
    float *dev_x;
    float *dev_y;
    
    cudaCheckReturn(hipMalloc((void **) &dev_A, (1300) * (1300) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (1300) * (1300) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_tmp, (1300) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_x, (1300) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_y, (1300) * sizeof(float)));
    
    
    cudaCheckReturn(hipMemcpy(dev_A, A, (1300) * (1300) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_B, B, (1300) * (1300) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_x, x, (1300) * sizeof(float), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(41);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, alpha, beta, dev_tmp, dev_x, dev_y);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(tmp, dev_tmp, (1300) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(y, dev_y, (1300) * sizeof(float), hipMemcpyDeviceToHost));
    
    
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_tmp));
    cudaCheckReturn(hipFree(dev_x));
    cudaCheckReturn(hipFree(dev_y));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;

  /* Variable declaration/allocation. */
  DATA_TYPE alpha;
  DATA_TYPE beta;
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, N, n, n);
  POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, N, N, n, n);
  POLYBENCH_1D_ARRAY_DECL(tmp, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(x, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(y, DATA_TYPE, N, n);


  /* Initialize array(s). */
  init_array (n, &alpha, &beta,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(B),
	      POLYBENCH_ARRAY(x));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_gesummv (n, alpha, beta,
		  POLYBENCH_ARRAY(A),
		  POLYBENCH_ARRAY(B),
		  POLYBENCH_ARRAY(tmp),
		  POLYBENCH_ARRAY(x),
		  POLYBENCH_ARRAY(y));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(y)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);
  POLYBENCH_FREE_ARRAY(tmp);
  POLYBENCH_FREE_ARRAY(x);
  POLYBENCH_FREE_ARRAY(y);

  return 0;
}
