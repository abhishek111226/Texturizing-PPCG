#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "trmm_kernel.hu"
__global__ void kernel0(float B[1000][1200])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c1 = 0; c1 <= 998; c1 += 32)
      for (int c2 = c1; c2 <= 999; c2 += 32) {
        if (32 * b0 + t0 <= 1199)
          for (int c4 = 0; c4 <= ppcg_min(ppcg_min(31, -c1 + 998), -c1 + c2 + 30); c4 += 1)
            for (int c5 = ppcg_max(0, c1 - c2 + c4 + 1); c5 <= ppcg_min(31, -c2 + 999); c5 += 1)
              B[c1 + c4][32 * b0 + t0] += ((tex2D(texRef_A, c1 + c4, c2 + c5)) * B[c2 + c5][32 * b0 + t0]);
        __syncthreads();
      }
}
__global__ void kernel1(float B[1000][1200], float alpha)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    if (32 * b0 + t0 <= 999)
      for (int c3 = t1; c3 <= ppcg_min(31, -32 * b1 + 1199); c3 += 16)
        B[32 * b0 + t0][32 * b1 + c3] = (alpha * B[32 * b0 + t0][32 * b1 + c3]);
}
