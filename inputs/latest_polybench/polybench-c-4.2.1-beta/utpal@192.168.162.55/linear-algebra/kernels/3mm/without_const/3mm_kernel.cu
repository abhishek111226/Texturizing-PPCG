#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "3mm_kernel.hu"
__global__ void kernel0(float A[800][1000], float B[1000][900], float E[800][900])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 999; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 899); c4 += 16) {
        if (c2 == 0)
          E[32 * b0 + t0][32 * b1 + c4] = 0.F;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 999); c5 += 1)
          E[32 * b0 + t0][32 * b1 + c4] += (A[32 * b0 + t0][c2 + c5] * B[c2 + c5][32 * b1 + c4]);
      }
      __syncthreads();
    }
}
__global__ void kernel1(float C[900][1200], float D[1200][1100], float F[900][1100])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 1199; c2 += 32) {
      if (32 * b0 + t0 <= 899)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16) {
          if (c2 == 0)
            F[32 * b0 + t0][32 * b1 + c4] = 0.F;
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 1199); c5 += 1)
            F[32 * b0 + t0][32 * b1 + c4] += (C[32 * b0 + t0][c2 + c5] * D[c2 + c5][32 * b1 + c4]);
        }
      __syncthreads();
    }
}
__global__ void kernel2(float E[800][900], float F[900][1100], float G[800][1100])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 899; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1099); c4 += 16) {
        if (c2 == 0)
          G[32 * b0 + t0][32 * b1 + c4] = 0.F;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 899); c5 += 1)
          G[32 * b0 + t0][32 * b1 + c4] += (E[32 * b0 + t0][c2 + c5] * F[c2 + c5][32 * b1 + c4]);
      }
      __syncthreads();
    }
}
