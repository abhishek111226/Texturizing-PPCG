#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "2mm_kernel.hu"
__global__ void kernel0(float A[800][1100], float alpha, float tmp[800][900])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 1099; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 899); c4 += 16) {
        if (c2 == 0)
          tmp[32 * b0 + t0][32 * b1 + c4] = 0.F;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 1099); c5 += 1)
          tmp[32 * b0 + t0][32 * b1 + c4] += ((alpha * A[32 * b0 + t0][c2 + c5]) * (tex2D(texRef_B, 32 * b1 + c4, c2 + c5)));
      }
      __syncthreads();
    }
}
__global__ void kernel1(float D[800][1200], float beta, float tmp[800][900])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 899; c2 += 32) {
      for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1199); c4 += 16) {
        if (c2 == 0)
          D[32 * b0 + t0][32 * b1 + c4] *= beta;
        for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 899); c5 += 1)
          D[32 * b0 + t0][32 * b1 + c4] += (tmp[32 * b0 + t0][c2 + c5] * (tex2D(texRef_C, 32 * b1 + c4, c2 + c5)));
      }
      __syncthreads();
    }
}
