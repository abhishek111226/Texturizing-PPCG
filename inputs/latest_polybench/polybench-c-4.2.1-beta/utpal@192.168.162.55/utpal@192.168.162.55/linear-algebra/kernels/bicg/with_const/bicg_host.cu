#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "bicg_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* bicg.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
extern "C"{
#include <polybench.h>
}
/* Include benchmark-specific header. */
#include "bicg.h"


/* Array initialization. */
static
void init_array (int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m))
{
  int i, j;

  for (i = 0; i < m; i++)
    p[i] = (DATA_TYPE)(i % m) / m;
  for (i = 0; i < n; i++) {
    r[i] = (DATA_TYPE)(i % n) / n;
    for (j = 0; j < m; j++)
      A[i][j] = (DATA_TYPE) (i*(j+1) % n)/n;
  }
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int m, int n,
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n))

{
  int i;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("s");
  for (i = 0; i < m; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, s[i]);
  }
  POLYBENCH_DUMP_END("s");
  POLYBENCH_DUMP_BEGIN("q");
  for (i = 0; i < n; i++) {
    if (i % 20 == 0) fprintf (POLYBENCH_DUMP_TARGET, "\n");
    fprintf (POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, q[i]);
  }
  POLYBENCH_DUMP_END("q");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_bicg(int m, int n,
		 DATA_TYPE POLYBENCH_2D(A,N,M,n,m),
		 DATA_TYPE POLYBENCH_1D(s,M,m),
		 DATA_TYPE POLYBENCH_1D(q,N,n),
		 DATA_TYPE POLYBENCH_1D(p,M,m),
		 DATA_TYPE POLYBENCH_1D(r,N,n))
{
  int i, j;

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    float *dev_p;
    float *dev_q;
    float *dev_r;
    float *dev_s;
    
    cudaCheckReturn(hipMalloc((void **) &dev_p, (1900) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_q, (2100) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_r, (2100) * sizeof(float)));
    cudaCheckReturn(hipMalloc((void **) &dev_s, (1900) * sizeof(float)));
    
    
    hipArray* cuArr_A;
    hipChannelFormatDesc channelDesc_A= hipCreateChannelDesc<float>();
    hipMallocArray(&cuArr_A, &channelDesc_A, 1900, 2100);
    hipMemcpyToArray(cuArr_A, 0, 0, A, (2100) * (1900) * sizeof(float), hipMemcpyHostToDevice);
    hipBindTextureToArray(texRef_A, cuArr_A, channelDesc_A);
    
    cudaCheckReturn(hipMemcpy(dev_p, p, (1900) * sizeof(float), hipMemcpyHostToDevice));
    cudaCheckReturn(hipMemcpy(dev_r, r, (2100) * sizeof(float), hipMemcpyHostToDevice));
    {
      dim3 k0_dimBlock(32);
      dim3 k0_dimGrid(60);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_r, dev_s);
      cudaCheckKernel();
    }
    
    {
      dim3 k1_dimBlock(32);
      dim3 k1_dimGrid(66);
      kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_p, dev_q);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(hipMemcpy(q, dev_q, (2100) * sizeof(float), hipMemcpyDeviceToHost));
    cudaCheckReturn(hipMemcpy(s, dev_s, (1900) * sizeof(float), hipMemcpyDeviceToHost));
    
    hipUnbindTexture(texRef_A);
    
    hipFreeArray(cuArr_A);
    cudaCheckReturn(hipFree(dev_p));
    cudaCheckReturn(hipFree(dev_q));
    cudaCheckReturn(hipFree(dev_r));
    cudaCheckReturn(hipFree(dev_s));
  }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int m = M;

  /* Variable declaration/allocation. */
  POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, N, M, n, m);
  POLYBENCH_1D_ARRAY_DECL(s, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(q, DATA_TYPE, N, n);
  POLYBENCH_1D_ARRAY_DECL(p, DATA_TYPE, M, m);
  POLYBENCH_1D_ARRAY_DECL(r, DATA_TYPE, N, n);

  /* Initialize array(s). */
  init_array (m, n,
	      POLYBENCH_ARRAY(A),
	      POLYBENCH_ARRAY(r),
	      POLYBENCH_ARRAY(p));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_bicg (m, n,
	       POLYBENCH_ARRAY(A),
	       POLYBENCH_ARRAY(s),
	       POLYBENCH_ARRAY(q),
	       POLYBENCH_ARRAY(p),
	       POLYBENCH_ARRAY(r));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(m, n, POLYBENCH_ARRAY(s), POLYBENCH_ARRAY(q)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(s);
  POLYBENCH_FREE_ARRAY(q);
  POLYBENCH_FREE_ARRAY(p);
  POLYBENCH_FREE_ARRAY(r);

  return 0;
}
