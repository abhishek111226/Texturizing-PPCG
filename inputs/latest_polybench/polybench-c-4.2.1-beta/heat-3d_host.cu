#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "heat-3d_kernel.hu"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* heat-3d.c: this file is part of PolyBench/C */

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
#include <polybench.h>

/* Include benchmark-specific header. */
#include "heat-3d.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n),
		 DATA_TYPE POLYBENCH_3D(B,N,N,N,n,n,n))
{
  int i, j, k;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      for (k = 0; k < n; k++)
        A[i][j][k] = B[i][j][k] = (DATA_TYPE) (i + j + (n-k))* 10 / (n);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n))

{
  int i, j, k;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("A");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      for (k = 0; k < n; k++) {
         if ((i * n * n + j * n + k) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
         fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, A[i][j][k]);
      }
  POLYBENCH_DUMP_END("A");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
static
void kernel_heat_3d(int tsteps,
		      int n,
		      DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n),
		      DATA_TYPE POLYBENCH_3D(B,N,N,N,n,n,n))
{
  int t, i, j, k;

    {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      float (*dev_A)[200][200];
      float (*dev_B)[200][200];
      
      cudaCheckReturn(hipMalloc((void **) &dev_A, (200) * (200) * (200) * sizeof(float)));
      cudaCheckReturn(hipMalloc((void **) &dev_B, (200) * (200) * (200) * sizeof(float)));
      
      
      cudaCheckReturn(hipMemcpy(dev_A, A, (200) * (200) * (200) * sizeof(float), hipMemcpyHostToDevice));
      cudaCheckReturn(hipMemcpy(dev_B, B, (200) * (200) * (200) * sizeof(float), hipMemcpyHostToDevice));
      for (int c0 = 1; c0 <= 1000; c0 += 1) {
        {
          dim3 k0_dimBlock(4, 4, 32);
          dim3 k0_dimGrid(7, 7);
          kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, c0);
          cudaCheckKernel();
        }
        
        {
          dim3 k1_dimBlock(4, 4, 32);
          dim3 k1_dimGrid(7, 7);
          kernel1 <<<k1_dimGrid, k1_dimBlock>>> (dev_A, dev_B, c0);
          cudaCheckKernel();
        }
        
      }
      cudaCheckReturn(hipMemcpy(A, dev_A, (200) * (200) * (200) * sizeof(float), hipMemcpyDeviceToHost));
      cudaCheckReturn(hipMemcpy(B, dev_B, (200) * (200) * (200) * sizeof(float), hipMemcpyDeviceToHost));
      
      
      cudaCheckReturn(hipFree(dev_A));
      cudaCheckReturn(hipFree(dev_B));
    }

}


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int tsteps = TSTEPS;

  /* Variable declaration/allocation. */
  POLYBENCH_3D_ARRAY_DECL(A, DATA_TYPE, N, N, N, n, n, n);
  POLYBENCH_3D_ARRAY_DECL(B, DATA_TYPE, N, N, N, n, n, n);


  /* Initialize array(s). */
  init_array (n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_heat_3d (tsteps, n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);

  return 0;
}
