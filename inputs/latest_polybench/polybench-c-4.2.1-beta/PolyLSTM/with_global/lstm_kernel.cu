#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "lstm_kernel.hu"
__global__ void kernel0(float c_F[400][2850], float *o, float s_F[ns <= 0 ? 399 : 400][2850], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < ns; c1 += 1048576)
      if (ns >= t0 + c1 + 1)
        s_F[c0 - 1][t0 + c1] = (c_F[c0 - 1][t0 + c1] * o[t0 + c1]);
}
__global__ void kernel1(float *i, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 2849)
      i[32 * b0 + t0] = 0.0;
}
__global__ void kernel2(float *f, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 2849)
      f[32 * b0 + t0] = 0.0;
}
__global__ void kernel3(float *g, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 2849)
      g[32 * b0 + t0] = 0.0;
}
__global__ void kernel4(float *o, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 2849)
      o[32 * b0 + t0] = 0.0;
}
__global__ void kernel5(float U_f[2850][3000], float *f, float inp_F[400][3000], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2999; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2999); c4 += 1)
          f[32 * b0 + t0] += (U_f[32 * b0 + t0][c2 + c4] * inp_F[c0][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel6(float U_g[2850][3000], float *g, float inp_F[400][3000], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2999; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2999); c4 += 1)
          g[32 * b0 + t0] += (U_g[32 * b0 + t0][c2 + c4] * inp_F[c0][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel7(float U_i[2850][3000], float *i, float inp_F[400][3000], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2999; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2999); c4 += 1)
          i[32 * b0 + t0] += (U_i[32 * b0 + t0][c2 + c4] * inp_F[c0][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel8(float W_f[2850][2850], float *f, float s_F[ns <= 0 ? 399 : 400][2850], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2849; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2849); c4 += 1)
          f[32 * b0 + t0] += (W_f[32 * b0 + t0][c2 + c4] * s_F[c0 - 1][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel9(float U_o[2850][3000], float inp_F[400][3000], float *o, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2999; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2999); c4 += 1)
          o[32 * b0 + t0] += (U_o[32 * b0 + t0][c2 + c4] * inp_F[c0][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel10(float W_i[2850][2850], float *i, float s_F[ns <= 0 ? 399 : 400][2850], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2849; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2849); c4 += 1)
          i[32 * b0 + t0] += (W_i[32 * b0 + t0][c2 + c4] * s_F[c0 - 1][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel11(float W_g[2850][2850], float *g, float s_F[ns <= 0 ? 399 : 400][2850], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2849; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2849); c4 += 1)
          g[32 * b0 + t0] += (W_g[32 * b0 + t0][c2 + c4] * s_F[c0 - 1][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel12(float W_o[2850][2850], float *o, float s_F[ns <= 0 ? 399 : 400][2850], int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 2849; c2 += 32) {
      if (32 * b0 + t0 <= 2849)
        for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 2849); c4 += 1)
          o[32 * b0 + t0] += (W_o[32 * b0 + t0][c2 + c4] * s_F[c0 - 1][c2 + c4]);
      __syncthreads();
    }
}
__global__ void kernel13(float c_F[400][2850], float *f, float *g, float *i, int ns, int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < ns; c1 += 1048576)
      if (ns >= t0 + c1 + 1)
        c_F[c0][t0 + c1] = ((c_F[c0 - 1][t0 + c1] * f[t0 + c1]) + (g[t0 + c1] * i[t0 + c1]));
}
