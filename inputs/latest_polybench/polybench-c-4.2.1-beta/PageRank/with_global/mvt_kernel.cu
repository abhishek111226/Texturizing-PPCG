#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "mvt_kernel.hu"
__global__ void kernel0(float A[4000][4000], float x1[4000], float x2[4000], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c2 = 0; c2 <= 3999; c2 += 32) {
      for (int c4 = 0; c4 <= 31; c4 += 1)
        x1[32 * b0 + t0] = (x1[32 * b0 + t0] + (A[32 * b0 + t0][c2 + c4] * x2[c2 + c4]));
      __syncthreads();
    }
}
__global__ void kernel1(float A[4000][4000], float x1[4000], float x2[4000], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c2 = 0; c2 <= 3999; c2 += 32) {
      for (int c4 = 0; c4 <= 31; c4 += 1)
        x2[32 * b0 + t0] = (x2[32 * b0 + t0] + (A[32 * b0 + t0][c2 + c4] * x1[c2 + c4]));
      __syncthreads();
    }
}
