#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "rnn_kernel.hu"
__global__ void kernel0(float U[2000][1000], float inp_F[1000][1000], float s_F[1000][2000])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_U[32][32];
    __shared__ float shared_inp_F[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c2 = 0; c2 <= 999; c2 += 32) {
      if (32 * b1 + t0 <= 1999)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 999); c4 += 16)
          shared_U[t0][c4] = U[32 * b1 + t0][c2 + c4];
      if (32 * b0 + t0 <= 999)
        for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 999); c4 += 16)
          shared_inp_F[t0][c4] = inp_F[32 * b0 + t0][c2 + c4];
      __syncthreads();
      if (32 * b0 + t0 <= 999)
        for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1999); c4 += 16)
          for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 999); c5 += 1)
            s_F[32 * b0 + t0][32 * b1 + c4] += (shared_U[c4][c5] * shared_inp_F[t0][c5]);
      __syncthreads();
    }
}
__global__ void kernel1(float W[2000][2000], float s_F[1000][2000], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;
    __shared__ float shared_W[32][32];
    __shared__ float shared_s_F_0[1][32];
    __shared__ float shared_s_F_1[1][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (32 * b0 + t0 <= 1999)
        shared_s_F_0[0][t0] = s_F[c0][32 * b0 + t0];
      for (int c2 = 0; c2 <= 1999; c2 += 32) {
        if (t0 + c2 <= 1999) {
          for (int c3 = 0; c3 <= ppcg_min(31, -32 * b0 + 1999); c3 += 1)
            shared_W[c3][t0] = W[32 * b0 + c3][t0 + c2];
          shared_s_F_1[0][t0] = s_F[c0 - 1][t0 + c2];
        }
        __syncthreads();
        if (32 * b0 + t0 <= 1999)
          for (int c4 = 0; c4 <= ppcg_min(31, -c2 + 1999); c4 += 1)
            shared_s_F_0[0][t0] += (shared_W[t0][c4] * shared_s_F_1[0][c4]);
        __syncthreads();
      }
      if (32 * b0 + t0 <= 1999)
        s_F[c0][32 * b0 + t0] = shared_s_F_0[0][t0];
    }
}
__global__ void kernel2(float V[1500][2000], float out_F[1000][1500], float s_F[1000][2000])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ float shared_V[32][32];
    __shared__ float shared_out_F[32][32];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (32 * b0 + t0 <= 999)
        for (int c1 = t1; c1 <= ppcg_min(31, -32 * b1 + 1499); c1 += 16)
          shared_out_F[t0][c1] = out_F[32 * b0 + t0][32 * b1 + c1];
      for (int c2 = 0; c2 <= 1999; c2 += 32) {
        if (32 * b1 + t0 <= 1499)
          for (int c4 = t1; c4 <= ppcg_min(31, -c2 + 1999); c4 += 16)
            shared_V[t0][c4] = V[32 * b1 + t0][c2 + c4];
        __syncthreads();
        if (32 * b0 + t0 <= 999)
          for (int c4 = t1; c4 <= ppcg_min(31, -32 * b1 + 1499); c4 += 16)
            for (int c5 = 0; c5 <= ppcg_min(31, -c2 + 1999); c5 += 1)
              shared_out_F[t0][c4] += (shared_V[c4][c5] * s_F[32 * b0 + t0][c2 + c5]);
        __syncthreads();
      }
      if (32 * b0 + t0 <= 999)
        for (int c1 = t1; c1 <= ppcg_min(31, -32 * b1 + 1499); c1 += 16)
          out_F[32 * b0 + t0][32 * b1 + c1] = shared_out_F[t0][c1];
    }
}
