#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "trisolv_kernel.hu"
__global__ void kernel0(float b[2000], float x[2000])
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (32 * b0 + t0 <= 1999)
      x[32 * b0 + t0] = b[32 * b0 + t0];
}
__global__ void kernel1(float x[2000], int c0)
{

    x[c0 / 2] = (x[c0 / 2] / (tex2D(texRef_L, c0 / 2, c0 / 2)));
}
__global__ void kernel2(float x[2000], int c0)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    if (c0 >= 64 * b0 + 2 * t0 + 1 && 32 * b0 + t0 + 1999 >= c0)
      x[-32 * b0 - t0 + c0] -= ((tex2D(texRef_L, 32 * b0 + t0, -32 * b0 - t0 + c0)) * x[32 * b0 + t0]);
}
