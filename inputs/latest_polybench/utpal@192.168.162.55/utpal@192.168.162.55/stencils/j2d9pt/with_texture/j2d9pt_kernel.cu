#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "j2d9pt_kernel.hu"
__global__ void kernel0(float B[999][1000])
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    if (32 * b0 + t0 >= 1 && 32 * b0 + t0 <= 998)
      for (int c3 = ppcg_max(t1, ((t1 + 15) % 16) - 32 * b1 + 1); c3 <= ppcg_min(31, -32 * b1 + 998); c3 += 16)
        B[32 * b0 + t0][32 * b1 + c3] = ((((((((((7 * (tex2D(texRef_A, 32 * b1 + c3 - 1, 32 * b0 + t0 - 1))) + (5 * (tex2D(texRef_A, 32 * b1 + c3, 32 * b0 + t0 - 1)))) + (9 * (tex2D(texRef_A, 32 * b1 + c3 + 1, 32 * b0 + t0 - 1)))) + (12 * (tex2D(texRef_A, 32 * b1 + c3 - 1, 32 * b0 + t0)))) + (15 * (tex2D(texRef_A, 32 * b1 + c3, 32 * b0 + t0)))) + (12 * (tex2D(texRef_A, 32 * b1 + c3 + 1, 32 * b0 + t0)))) + (9 * (tex2D(texRef_A, 32 * b1 + c3 - 1, 32 * b0 + t0 + 1)))) + (5 * (tex2D(texRef_A, 32 * b1 + c3, 32 * b0 + t0 + 1)))) + (7 * (tex2D(texRef_A, 32 * b1 + c3 + 1, 32 * b0 + t0 + 1)))) / 118);
}
