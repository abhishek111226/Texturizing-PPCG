#include "hip/hip_runtime.h"
#include <stdio.h> 
#define DEVICECODE true 
#include "doitgen_kernel.hu"
__global__ void kernel0(float sum[160], int c0, int c1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    sum[32 * b0 + t0] = 0.F;
}
__global__ void kernel1(float A[150][140][160], float C4[160][160], float sum[160], int c0, int c1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    for (int c3 = 0; c3 <= 159; c3 += 32) {
      for (int c5 = 0; c5 <= 31; c5 += 1)
        sum[32 * b0 + t0] += (A[c0][c1][c3 + c5] * C4[c3 + c5][32 * b0 + t0]);
      __syncthreads();
    }
}
__global__ void kernel2(float A[150][140][160], float sum[160], int c0, int c1)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    A[c0][c1][32 * b0 + t0] = sum[32 * b0 + t0];
}
