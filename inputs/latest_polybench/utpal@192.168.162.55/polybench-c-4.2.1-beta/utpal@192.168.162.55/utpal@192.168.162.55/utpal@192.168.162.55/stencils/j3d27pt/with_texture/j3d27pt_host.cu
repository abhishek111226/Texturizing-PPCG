#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "j3d27pt_kernel.hu"
/**
 * jacobi-2d-imper.c: This file is part of the PolyBench/C 3.2 test suite.
 *
 *
 * Contact: Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://polybench.sourceforge.net
 */
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>

/* Include polybench common header. */
extern "C"{
#include <polybench.h>
}
/* Include benchmark-specific header. */
/* Default data type is double, default size is 20x1000. */
#include "jacobi-3d-imper.h"


/* Array initialization. */
static
void init_array (int n,
		 DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n),
		 DATA_TYPE POLYBENCH_3D(B,N,N,N,n,n,n))
{
  int i, j, k;

  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      for (k = 0; k < n; k++)
        A[i][j][k] = B[i][j][k] = (DATA_TYPE) (i + j + (n-k))* 10 / (n);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int n,
		 DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n))

{
  int i, j, k;

  POLYBENCH_DUMP_START;
  POLYBENCH_DUMP_BEGIN("A");
  for (i = 0; i < n; i++)
    for (j = 0; j < n; j++)
      for (k = 0; k < n; k++) {
         if ((i * n * n + j * n + k) % 20 == 0) fprintf(POLYBENCH_DUMP_TARGET, "\n");
         fprintf(POLYBENCH_DUMP_TARGET, DATA_PRINTF_MODIFIER, A[i][j][k]);
      }
  POLYBENCH_DUMP_END("A");
  POLYBENCH_DUMP_FINISH;
}


/* Main computational kernel. The whole function will be timed,
   including the call and return. */
	 static
	 void kernel_heat_3d(int tsteps,
	 		      int n,
	 		      DATA_TYPE POLYBENCH_3D(A,N,N,N,n,n,n),
	 		      DATA_TYPE POLYBENCH_3D(B,N,N,N,n,n,n))
	 {
	   int t, i, j, k;

	 {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

	   float (*dev_B)[1000][1000];
	   
	   cudaCheckReturn(hipMalloc((void **) &dev_B, (999) * (1000) * (1000) * sizeof(float)));
	   
	   
	   hipArray* cuArr_A;
	   hipChannelFormatDesc channelDesc_A= hipCreateChannelDesc<float>();
	   const hipExtent extent_A= make_hipExtent(1000, 1000, 1000);
	   hipMalloc3DArray(&cuArr_A, &channelDesc_A, extent_A);
	   hipMemcpy3DParms copyParams_A_to_device= {0};
	   copyParams_A_to_device.srcPtr = make_hipPitchedPtr((void*)A, extent_A.width*sizeof(float), extent_A.width, extent_A.height);
	   copyParams_A_to_device.dstArray = cuArr_A;
	   copyParams_A_to_device.extent = extent_A;
	   copyParams_A_to_device.kind = hipMemcpyHostToDevice;
	   hipMemcpy3D(&copyParams_A_to_device);
	   hipBindTextureToArray(texRef_A, cuArr_A, channelDesc_A);
	   
	   cudaCheckReturn(hipMemcpy(dev_B, B, (999) * (1000) * (1000) * sizeof(float), hipMemcpyHostToDevice));
	   {
	     dim3 k0_dimBlock(4, 4, 32);
	     dim3 k0_dimGrid(32, 32);
	     kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_B);
	     cudaCheckKernel();
	   }
	   
	   cudaCheckReturn(hipMemcpy(B, dev_B, (999) * (1000) * (1000) * sizeof(float), hipMemcpyDeviceToHost));
	   
	   hipUnbindTexture(texRef_A);
	   
	   hipFreeArray(cuArr_A);
	   cudaCheckReturn(hipFree(dev_B));
	 }

	 }


int main(int argc, char** argv)
{
  /* Retrieve problem size. */
  int n = N;
  int tsteps = TSTEPS;

  /* Variable declaration/allocation. */
	POLYBENCH_3D_ARRAY_DECL(A, DATA_TYPE, N, N, N, n, n, n);
	POLYBENCH_3D_ARRAY_DECL(B, DATA_TYPE, N, N, N, n, n, n);


  /* Initialize array(s). */
  init_array (n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Start timer. */
  polybench_start_instruments;

  /* Run kernel. */
  kernel_heat_3d (tsteps, n, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B));

  /* Stop and print timer. */
  polybench_stop_instruments;
  polybench_print_instruments;

  /* Prevent dead-code elimination. All live-out data must be printed
     by the function call in argument. */
  polybench_prevent_dce(print_array(n, POLYBENCH_ARRAY(A)));

  /* Be clean. */
  POLYBENCH_FREE_ARRAY(A);
  POLYBENCH_FREE_ARRAY(B);

  return 0;
}
