#include <assert.h>
#include <stdio.h>
#define HOSTCODE true 
#include "3d_copy_kernel.hu"
//#include <stdio.h>
int A[300][300][300];
int B[300][300][300];
int main()
{
 int i,j,k; 	
 {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

   
   
   
   hipArray* cuArr_A;
   hipChannelFormatDesc channelDesc_A= hipCreateChannelDesc<int>();
   const hipExtent extent_A= make_hipExtent(300, 300, 299);
   hipMalloc3DArray(&cuArr_A, &channelDesc_A, extent_A);
   hipMemcpy3DParms copyParams_A_to_device= {0};
   copyParams_A_to_device.srcPtr = make_hipPitchedPtr((void*)A, extent_A.width*sizeof(int), extent_A.width, extent_A.height);
   copyParams_A_to_device.dstArray = cuArr_A;
   copyParams_A_to_device.extent = extent_A;
   copyParams_A_to_device.kind = hipMemcpyHostToDevice;
   hipMemcpy3D(&copyParams_A_to_device);
   hipBindTextureToArray(texRef_A, cuArr_A, channelDesc_A);
   
   hipArray* cuArr_B;
   hipChannelFormatDesc channelDesc_B= hipCreateChannelDesc<int>();
   const hipExtent extent_B= make_hipExtent(300, 300, 300);
   hipMalloc3DArray(&cuArr_B, &channelDesc_B, extent_B, hipArraySurfaceLoadStore);
   hipMemcpy3DParms copyParams_B_to_device= {0};
   copyParams_B_to_device.srcPtr = make_hipPitchedPtr((void*)B, extent_B.width*sizeof(int), extent_B.width, extent_B.height);
   copyParams_B_to_device.dstArray = cuArr_B;
   copyParams_B_to_device.extent = extent_B;
   copyParams_B_to_device.kind = hipMemcpyHostToDevice;
   hipMemcpy3D(&copyParams_B_to_device);
   cudaBindSurfaceToArray(surfRef_B, cuArr_B, channelDesc_B);
   
   {
     dim3 k0_dimBlock(4, 4, 32);
     dim3 k0_dimGrid(10, 10);
     kernel0 <<<k0_dimGrid, k0_dimBlock>>> ();
     cudaCheckKernel();
   }
   
   hipMemcpy3DParms copyParams_B_from_device= {0};
   copyParams_B_from_device.dstPtr = make_hipPitchedPtr((void*)B, extent_B.width*sizeof(int), extent_B.width, extent_B.height);
   copyParams_B_from_device.srcArray = cuArr_B;
   copyParams_B_from_device.extent = extent_B;
   copyParams_B_from_device.kind = hipMemcpyDeviceToHost;
   hipMemcpy3D(&copyParams_B_from_device);
   
   
   hipUnbindTexture(texRef_A);
   
   hipFreeArray(cuArr_A);
   hipFreeArray(cuArr_B);
 }
  return 0;
}

